#include "hip/hip_runtime.h"
#pragma once

#include "CudaNbody.cuh"

using namespace std;



__global__ void getGrav(float4*cu_pos, float4 *cu_gravs)
{
	__shared__ float4 sharedPos[1024 * 2];
	//the pos of the current thread
	float4 myPosition;

	int i;
	
	//result to be copied back to host
	float4 result = { 0.0f, 0.0f, 0.0f, 0.0f };

	//thread id
	int gtid;
	gtid = ( blockIdx.x * blockDim.x ) + threadIdx.x;


	myPosition = cu_pos[gtid];
	sharedPos[gtid] = cu_pos[gtid];

	for (i = 0; i < 1024 * 2; i ++)
	{
		
			//body body interaction part
			float3 delta;

			delta.x = sharedPos[i].x - myPosition.x;
			delta.y = sharedPos[i].y - myPosition.y;
			delta.z = sharedPos[i].z - myPosition.z;

			float distance = sqrt(delta.x * delta.x + delta.y * delta.y + delta.z * delta.z) ;

			if (distance > 0.1f)
			{
				//direction of the gravity force
				float4 dir = { delta.x / distance, delta.y / distance, delta.z / distance, 0.0f };
				//size of the gravity force F = G * m1*m2/d^2
				result.x += ((float)6.67408* (sharedPos[i].w * myPosition.w) / (distance * distance)) * dir.x;
				result.y += ((float)6.67408* (sharedPos[i].w * myPosition.w) / (distance * distance)) * dir.y;
				result.z += ((float)6.67408* (sharedPos[i].w * myPosition.w) / (distance * distance)) * dir.z;
				result.w = 0.0f;
				//printf("distance -> %f", distance);
				__syncthreads();
			}

	}
	cu_gravs[gtid] = result;

	
}

void Cuda::getGravities( std::vector<glm::vec3>& gravs, int BODIES, float dt)
{
	auto grav_size = sizeof(float4) * BODIES;

	float4 gravList[1024 * 2];

	//call the kernel
	getGrav <<<BODIES / 512, 512 >>> (positionBuff, gravityBuff);

	// Wait for kernel to complete check for errors
	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	//get the data back from the buffer 
	auto temp = hipMemcpy(&gravList[0], gravityBuff, grav_size, hipMemcpyDeviceToHost);

	for (int i = 0; i < BODIES; i++)
	{
		gravs[i].x = gravList[i].x;
		gravs[i].y = gravList[i].y;
		gravs[i].z = gravList[i].z;
	}

	hipFree(positionBuff);
	hipFree(gravityBuff);
	hipFree(velocityBuf);
	hipFree(dtBuf);

	
}


void  Cuda::loadBuffers(int BODIES, std::vector<Body*> bodyList, std::vector<glm::vec3>& gravs, float dt)
{
	//data size
	auto posList_size = sizeof(float4) * BODIES;
	auto grav_size = sizeof(float4) * BODIES;
	auto vel_size = sizeof(float4) * BODIES;
	auto dt_size = sizeof(float);

	//lists
	float4 posLis[1024 * 2];
	float4 gravList[1024 * 2];
	float4 velList[1024 * 2];


	//Prepare data for GPU
	


	//populate position list nad grav list
	for (int i = 0; i < BODIES; i++)
	{
		posLis[i] = { bodyList[i]->getPos().x,bodyList[i]->getPos().y,bodyList[i]->getPos().z, bodyList[i]->getMass() };	
	}

	//allocate memory
	hipMalloc((void**)&positionBuff, posList_size);
	hipMalloc((void**)&gravityBuff, grav_size);
	

	
	//copy data to GPU
	hipError_t s1 = hipMemcpy(positionBuff, &posLis[0], posList_size, hipMemcpyHostToDevice);
	if (s1 != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(s1));

	hipError_t s2 = hipMemcpy(gravityBuff, &gravList[0], grav_size, hipMemcpyHostToDevice);
	if (s2 != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(s2));
	
	

}
