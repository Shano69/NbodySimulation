#include "hip/hip_runtime.h"
#pragma once

#include "CudaNbody.cuh"

using namespace std;



__global__ void getGrav(float4*cu_pos, float4 *cu_gravs, float4 *cu_vel, float dt)
{
	
	//the pos of the current thread
	

	int i;
	
	//result to be copied back to host
	float4 result = { 0.0f, 0.0f, 0.0f, 0.0f };

	//thread id
	int gtid;
	gtid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	for (i = 0; i < 1024 * 2; i ++)
	{
		
			//body body interaction part
			float3 delta;

			delta.x = cu_pos[i].x - cu_pos[gtid].x;
			delta.y = cu_pos[i].y - cu_pos[gtid].y;
			delta.z = cu_pos[i].z - cu_pos[gtid].z;

			float distance = sqrt(delta.x * delta.x + delta.y * delta.y + delta.z * delta.z) ;

			if (distance > 0.1f)
			{
				//direction of the gravity force
				float4 dir = { delta.x / distance, delta.y / distance, delta.z / distance, 0.0f };
				//size of the gravity force F = G * m1*m2/d^2
				result.x += ((float)6.67408* (cu_pos[i].w * cu_pos[gtid].w) / (distance * distance)) * dir.x;
				result.y += ((float)6.67408* (cu_pos[i].w * cu_pos[gtid].w) / (distance * distance)) * dir.y;
				result.z += ((float)6.67408* (cu_pos[i].w * cu_pos[gtid].w) / (distance * distance)) * dir.z;
				result.w = 0.0f;
				//printf("distance -> %f", distance);
				__syncthreads();
			}

	}

		cu_gravs[gtid] = result;
}

void Cuda::getGravities( std::vector<glm::vec3>& gravs, int BODIES, float dt)
{
	auto grav_size = sizeof(float4) * BODIES;

	float4 gravList[1024 * 2];

	//call the kernel
	getGrav <<<BODIES / 512, 512 >>> (positionBuff, gravityBuff, velocityBuf, dt);

	// Wait for kernel to complete check for errors
	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));

	//get the data back from the buffer 
	auto temp = hipMemcpy(&gravList[0], gravityBuff, grav_size, hipMemcpyDeviceToHost);

	for (int i = 0; i < BODIES; i++)
	{
		gravs[i].x = gravList[i].x;
		gravs[i].y = gravList[i].y;
		gravs[i].z = gravList[i].z;
	}

	hipFree(positionBuff);
	hipFree(gravityBuff);
	hipFree(velocityBuf);
	hipFree(dtBuf);
}


void  Cuda::loadBuffers(int BODIES, std::vector<Body*> bodyList, std::vector<glm::vec3>& gravs, float dt)
{
	//data size
	auto posList_size = sizeof(float4) * BODIES;
	auto grav_size = sizeof(float4) * BODIES;
	auto vel_size = sizeof(float4) * BODIES;
	auto dt_size = sizeof(float);

	//lists
	float4 posLis[1024 * 2];
	float4 gravList[1024 * 2];
	float4 velList[1024 * 2];


	//Prepare data for GPU
	


	//populate position list nad grav list
	for (int i = 0; i < BODIES; i++)
	{
		gravList[i] = { bodyList[i]->getAcc().x,bodyList[i]->getAcc().y,bodyList[i]->getAcc().z, 0.0f };
		posLis[i] = { bodyList[i]->getPos().x,bodyList[i]->getPos().y,bodyList[i]->getPos().z, bodyList[i]->getMass() };
		velList[i] = { bodyList[i]->getVel().x,bodyList[i]->getVel().y,bodyList[i]->getVel().z, 0.0f };
	}

	//allocate memory
	hipMalloc((void**)&positionBuff, posList_size);
	hipMalloc((void**)&gravityBuff, grav_size);
	hipMalloc((void**)&velocityBuf, vel_size);
	hipMalloc((void**)&dtBuf, dt_size);

	
	//copy data to GPU
	hipError_t s1 = hipMemcpy(positionBuff, &posLis[0], posList_size, hipMemcpyHostToDevice);
	if (s1 != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(s1));

	hipError_t s2 = hipMemcpy(gravityBuff, &gravList[0], grav_size, hipMemcpyHostToDevice);
	if (s2 != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(s2));

	hipError_t s3 = hipMemcpy(velocityBuf, &velList[0], posList_size, hipMemcpyHostToDevice);
	if (s3 != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(s3));

	hipError_t s4 = hipMemcpy(dtBuf, &dt, dt_size, hipMemcpyHostToDevice);
	if (s4 != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(s4));
	
	


}
